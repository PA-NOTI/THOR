#include "hip/hip_runtime.h"
// ==============================================================================
// This file is part of THOR.
//
//     THOR is free software : you can redistribute it and / or modify
//     it under the terms of the GNU General Public License as published by
//     the Free Software Foundation, either version 3 of the License, or
//     (at your option) any later version.
//
//     THOR is distributed in the hope that it will be useful,
//     but WITHOUT ANY WARRANTY; without even the implied warranty of
//     MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the
//     GNU General Public License for more details.
//
//     You find a copy of the GNU General Public License in the main
//     THOR directory under <license.txt>.If not, see
//     <http://www.gnu.org/licenses/>.
// ==============================================================================
//
// ESP -  Exoclimes Simulation Platform. (version 1.0)
//
//
//
// Method: Boundary layer (surface friction) physics module
//
//
// Known limitations: - Runs in a single GPU.
//
// Known issues: None
//
//
// If you use this code please cite the following reference:
//
//       [1] Mendonca, J.M., Grimm, S.L., Grosheintz, L., & Heng, K., ApJ, 829, 115, 2016
//
// Current Code Owner: Joao Mendonca, EEG. joao.mendonca@csh.unibe.ch
//                     Russell Deitrick (russell.deitrick@csh.unibe.ch)
//                     Urs Schroffenegger (urs.schroffenegger@csh.unibe.ch)
//
// History:
// Version Date       Comment
// ======= ====       =======
//
// 1.0     16/08/2017 Released version  (JM)
//
////////////////////////////////////////////////////////////////////////
#include "boundary_layer.h"

boundary_layer::boundary_layer() : bl_type_str("RayleighHS") {
}

boundary_layer::~boundary_layer() {
}

void boundary_layer::print_config() {
    log::printf("  Boundary layer module\n");

    // basic properties
    log::printf("    bl_type                    = %s \n", bl_type_str.c_str());
    log::printf("    surf_drag                  = %e 1/s\n", surf_drag_config);
    log::printf("    bl_sigma                   = %f \n", bl_sigma_config);

    log::printf("\n");
}

bool boundary_layer::initialise_memory(const ESP &              esp,
                                       device_RK_array_manager &phy_modules_core_arrays) {

    // hipMalloc((void **)&dvdz_tmp, 3 * esp.nvi * esp.point_num * sizeof(double));
    hipMalloc((void **)&d2vdz2_tmp, 3 * esp.nv * esp.point_num * sizeof(double));

    hipMalloc((void **)&atmp, esp.nv * esp.point_num * sizeof(double));
    hipMalloc((void **)&btmp, esp.nv * esp.point_num * sizeof(double));
    hipMalloc((void **)&ctmp, esp.nv * esp.point_num * sizeof(double));
    hipMalloc((void **)&cpr_tmp, esp.nv * esp.point_num * sizeof(double));
    hipMalloc((void **)&dtmp, 3 * esp.nv * esp.point_num * sizeof(double));
    hipMalloc((void **)&dpr_tmp, 3 * esp.nv * esp.point_num * sizeof(double));
    hipMalloc((void **)&RiB_d, esp.nvi * esp.point_num * sizeof(double));
    hipMalloc((void **)&KM_d, esp.nvi * esp.point_num * sizeof(double));
    hipMalloc((void **)&KH_d, esp.nvi * esp.point_num * sizeof(double));

    hipMalloc((void **)&bl_top_lev_d, esp.point_num * sizeof(int));

    RiB_h        = (double *)malloc(esp.nvi * esp.point_num * sizeof(double));
    KM_h         = (double *)malloc(esp.nvi * esp.point_num * sizeof(double));
    KH_h         = (double *)malloc(esp.nvi * esp.point_num * sizeof(double));
    bl_top_lev_h = (int *)malloc(esp.point_num * sizeof(int));

    hipMemset(atmp, 0, sizeof(double) * esp.point_num * esp.nv);
    hipMemset(btmp, 0, sizeof(double) * esp.point_num * esp.nv);
    hipMemset(ctmp, 0, sizeof(double) * esp.point_num * esp.nv);
    hipMemset(cpr_tmp, 0, sizeof(double) * esp.point_num * esp.nv);
    hipMemset(dtmp, 0, sizeof(double) * 3 * esp.point_num * esp.nv);
    hipMemset(dpr_tmp, 0, sizeof(double) * 3 * esp.point_num * esp.nv);
    hipMemset(RiB_d, 0, sizeof(double) * esp.point_num * esp.nvi);
    hipMemset(KM_d, 0, sizeof(double) * esp.point_num * esp.nvi);
    hipMemset(KH_d, 0, sizeof(double) * esp.point_num * esp.nvi);
    hipMemset(bl_top_lev_d, 0, sizeof(int) * esp.point_num);

    return true;
}


bool boundary_layer::free_memory() {

    // hipFree(dvdz_tmp);
    hipFree(d2vdz2_tmp);

    return true;
}

bool boundary_layer::initial_conditions(const ESP &esp, const SimulationSetup &sim, storage *s) {
    bool config_OK = true;

    bl_type = RAYLEIGHHS;
    if (bl_type_str == "RayleighHS") {
        bl_type = RAYLEIGHHS;
        config_OK &= true;
    }
    else if (bl_type_str == "MoninObukhov" || bl_type_str == "MO") {
        bl_type = MONINOBUKHOV;
        config_OK &= true;
    }
    else if (bl_type_str == "EkmanSpiral" || bl_type_str == "Ekman") {
        bl_type = EKMANSPIRAL;
        config_OK &= true;
    }
    else {
        log::printf("bl_type config item not recognised: [%s]\n", bl_type_str.c_str());
        config_OK &= false;
    }

    if (!config_OK) {
        log::printf("Error in configuration file\n");
        exit(-1);
    }

    BLSetup(esp, sim, bl_type, surf_drag_config, bl_sigma_config);

    return true;
}

bool boundary_layer::phy_loop(ESP &                  esp,
                              const SimulationSetup &sim,
                              int                    nstep, // Step number
                              double                 time_step) {           // Time-step [s]

    //  Number of threads per block.
    const int NTH = 256;

    //  Specify the block sizes.
    dim3 NB((esp.point_num / NTH) + 1, esp.nv, 1);
    dim3 NBLEV((esp.point_num / NTH) + 1, 1, 1);

    if (bl_type == RAYLEIGHHS) {
        rayleighHS<<<NB, NTH>>>(esp.Mh_d,
                                esp.pressure_d,
                                esp.Rho_d,
                                esp.Altitude_d,
                                surf_drag,
                                bl_sigma,
                                sim.Gravit,
                                time_step,
                                esp.point_num);
    }
    else if (bl_type == MONINOBUKHOV) {
        printf("MO BL not ready yet!\n");
    }
    else if (bl_type == EKMANSPIRAL) {
        // hipMemset(dvdz_tmp, 0, sizeof(double) * 3 * esp.point_num * esp.nvi);
        hipMemset(d2vdz2_tmp, 0, sizeof(double) * 3 * esp.point_num * esp.nv);

        // ConstKMEkman<<<NBLEV, NTH>>>(esp.Mh_d,
        //                              esp.pressure_d,
        //                              esp.Rho_d,
        //                              esp.Altitude_d,
        //                              esp.Altitudeh_d,
        //                              d2vdz2_tmp,
        //                              KMconst,
        //                              zbl,
        //                              time_step,
        //                              esp.point_num,
        //                              esp.nv);

        CalcRiB<<<NBLEV, NTH>>>(esp.pressure_d,
                                esp.Rho_d,
                                esp.Mh_d,
                                esp.Tsurface_d,
                                esp.Altitude_d,
                                esp.Altitudeh_d,
                                sim.Rd,
                                sim.Cp,
                                sim.P_Ref,
                                sim.Gravit,
                                RiB_d,
                                esp.point_num,
                                esp.nv);

        // TO DO
        // need KM array, KH array, general thomas solver for KM, KH
        // calc BL height from RiB
        // adjust Tsurface for sensible heat flux
        // how to adjust pressure? adjust pt first, then compute pressure? or is there a shortcut?
        // update pressure (implicitly) here, or add to qheat?

        MomentumDiff_Impl<<<NBLEV, NTH>>>(esp.Mh_d,
                                          esp.pressure_d,
                                          esp.Rho_d,
                                          esp.Altitude_d,
                                          esp.Altitudeh_d,
                                          atmp,
                                          btmp,
                                          ctmp,
                                          cpr_tmp,
                                          dtmp,
                                          dpr_tmp,
                                          KM_d,
                                          zbl,
                                          time_step,
                                          esp.point_num,
                                          esp.nv,
                                          bl_top_lev_d);
    }


    return true;
}

bool boundary_layer::configure(config_file &config_reader) {

    config_reader.append_config_var("bl_type", bl_type_str, string(bl_type_default)); //

    // coefficient of drag strength
    config_reader.append_config_var("surf_drag", surf_drag_config, surf_drag_config);

    // percent of surface pressure where bl starts
    config_reader.append_config_var("bl_sigma", bl_sigma_config, bl_sigma_config);

    return true;
}

bool boundary_layer::store(const ESP &esp, storage &s) {

    return true;
}

bool boundary_layer::store_init(storage &s) {
    s.append_value(surf_drag, "/surf_drag", "1/s", "surface drag coefficient");
    s.append_value(bl_sigma, "/bl_sigma", " ", "boundary layer sigma coordinate");

    return true;
}

void boundary_layer::BLSetup(const ESP &            esp,
                             const SimulationSetup &sim,
                             int                    bl_type,
                             double                 surf_drag_,
                             double                 bl_sigma_) {
    if (bl_type == RAYLEIGHHS) {
        surf_drag = surf_drag_;
        bl_sigma  = bl_sigma_;
    }
    else if (bl_type == EKMANSPIRAL) {
        double KMconst = 12.5;

        zbl = bl_sigma_ * sim.Top_altitude;

        for (int id = 0; id < esp.point_num; id++) {
            int lev = 0;
            while (esp.Altitude_h[lev] < zbl) {
                bl_top_lev_h[id] = lev;
                lev++;
            }

            for (lev = 0; lev < esp.nvi; lev++) {
                KM_h[id * esp.nvi + lev] = KMconst;
            }
        }
        hipMemcpy(KM_d, KM_h, esp.nvi * esp.point_num * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(bl_top_lev_d, bl_top_lev_h, esp.point_num * sizeof(int), hipMemcpyHostToDevice);
    }
    // printf("%f, %f, %d\n", zbl, esp.Altitude_h[bl_top_lev], bl_top_lev);
}


__global__ void rayleighHS(double *Mh_d,
                           double *pressure_d,
                           double *Rho_d,
                           double *Altitude_d,
                           double  surf_drag,
                           double  bl_sigma,
                           double  Gravit,
                           double  time_step,
                           int     num) {

    int id  = blockIdx.x * blockDim.x + threadIdx.x;
    int nv  = gridDim.y;
    int lev = blockIdx.y;

    if (id < num) {
        double sigma;
        double sigmab = bl_sigma;
        double kf     = surf_drag;
        double kv_hs;
        double ps, pre;
        double psm1;

        //      Calculates surface pressure
        psm1 = pressure_d[id * nv + 1]
               - Rho_d[id * nv + 0] * Gravit * (-Altitude_d[0] - Altitude_d[1]);
        ps = 0.5 * (pressure_d[id * nv + 0] + psm1);

        pre   = pressure_d[id * nv + lev];
        sigma = (pre / ps);

        //      Momentum dissipation constant.
        kv_hs = kf * max(0.0, (sigma - sigmab) / (1.0 - sigmab));

        //      Update momenta
        for (int k = 0; k < 3; k++)
            Mh_d[id * 3 * nv + lev * 3 + k] =
                Mh_d[id * 3 * nv + lev * 3 + k] / (1.0 + kv_hs * time_step);

        // Wh_d[id * (nv + 1) + lev + k] = Wh_d[id * (nv + 1) + lev + k] / (1.0 + kv_hs * time_step);
    }
}

__global__ void ConstKMEkman(double *Mh_d,
                             double *pressure_d,
                             double *Rho_d,
                             double *Altitude_d,
                             double *Altitudeh_d,
                             double *d2vdz2_tmp,
                             double  KMconst,
                             double  zbl,
                             double  time_step,
                             int     num,
                             int     nv) {

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int lev;

    if (id < num) {
        for (int k = 0; k < 3; k++) {
            // dvdz_tmp[id * 3 * (nv + 1) + 0 * 3 + k]  = 0; //boundary condition
            // dvdz_tmp[id * 3 * (nv + 1) + nv * 3 + k] = 0;
            // for (lev = 1; lev < nv; lev++) {
            //     //first derivative at interfaces (half-layers)
            //     dvdz_tmp[id * 3 * (nv + 1) + lev * 3 + k] =
            //         (Mh_d[id * 3 * nv + lev * 3 + k] / Rho_d[id * nv + lev]
            //          - Mh_d[id * 3 * nv + (lev - 1) * 3 + k] / Rho_d[id * nv + lev - 1])
            //         / (Altitude_d[lev] - Altitude_d[lev - 1]);
            // }
            // for (lev = 0; lev < nv; lev++) {
            //     d2vdz2_tmp[id * 3 * nv + lev * 3 + k] = (dvdz_tmp[id * 3 * nv + (lev + 1) * 3 + k]
            //                                              - dvdz_tmp[id * 3 * nv + lev * 3 + k])
            //                                             / (Altitudeh_d[lev + 1] - Altitudeh_d[lev]);
            //     Mh_d[id * 3 * nv + lev * 3 + k] += -Rho_d[id * nv + lev] * KMconst
            //                                        * d2vdz2_tmp[id * 3 * nv + lev * 3 + k]
            //                                        * time_step;
            // }
            for (lev = 0; lev < nv; lev++) {
                if (Altitude_d[lev] < zbl) {
                    if (lev == 0) { //lowest layer, v at lowest boundary = 0, dz0 = Altitude0
                        d2vdz2_tmp[id * 3 * nv + lev * 3 + k] =
                            ((Mh_d[id * 3 * nv + (lev + 1) * 3 + k]
                              - Mh_d[id * 3 * nv + (lev)*3 + k])
                                 / (Altitude_d[lev + 1] - Altitude_d[lev])
                             - (Mh_d[id * 3 * nv + (lev)*3 + k]) / (Altitude_d[lev]))
                            / (Altitudeh_d[lev + 1] - Altitudeh_d[lev]);
                    }
                    // else if (lev == nv - 1) { //top layer,
                    //     ((Mh_d[id * 3 * nv + (lev + 1) * 3 + k] / Rho_d[id * nv + lev + 1]
                    //       - Mh_d[id * 3 * nv + (lev)*3 + k] / Rho_d[id * nv + lev])
                    //          / (Altitude_d[lev + 1] - Altitude_d[lev])
                    //      - (Mh_d[id * 3 * nv + (lev)*3 + k] / Rho_d[id * nv + lev]
                    //         - Mh_d[id * 3 * nv + (lev - 1) * 3 + k] / Rho_d[id * nv + lev - 1])
                    //            / (Altitude_d[lev] - Altitude_d[lev - 1]))
                    //         / (Altitudeh_d[lev + 1] - Altitudeh_d[lev]);
                    // }
                    else { //might need to add a term to layer above to conserve momentum
                        d2vdz2_tmp[id * 3 * nv + lev * 3 + k] =
                            ((Mh_d[id * 3 * nv + (lev + 1) * 3 + k]
                              - Mh_d[id * 3 * nv + (lev)*3 + k])
                                 / (Altitude_d[lev + 1] - Altitude_d[lev])
                             - (Mh_d[id * 3 * nv + (lev)*3 + k]
                                - Mh_d[id * 3 * nv + (lev - 1) * 3 + k])
                                   / (Altitude_d[lev] - Altitude_d[lev - 1]))
                            / (Altitudeh_d[lev + 1] - Altitudeh_d[lev]);
                    }
                    Mh_d[id * 3 * nv + lev * 3 + k] +=
                        KMconst * d2vdz2_tmp[id * 3 * nv + lev * 3 + k] * time_step;
                }
            }
        }
    }
}

__global__ void MomentumDiff_Impl(double *Mh_d,
                                  double *pressure_d,
                                  double *Rho_d,
                                  double *Altitude_d,
                                  double *Altitudeh_d,
                                  double *atmp,
                                  double *btmp,
                                  double *ctmp,
                                  double *cpr_tmp,
                                  double *dtmp,
                                  double *dpr_tmp,
                                  double *KM_d,
                                  double  zbl,
                                  double  time_step,
                                  int     num,
                                  int     nv,
                                  int *   bl_top_lev_d) {

    //should create check on stability of thomas algorithm

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int lev;

    if (id < num) {
        for (lev = 0; lev < bl_top_lev_d[id] + 1; lev++) {
            //forward sweep
            if (lev == 0) { //lowest layer, v at lowest boundary = 0, dz0 = Altitude0
                atmp[id * nv + lev] = 0;
                btmp[id * nv + lev] =
                    -(1.0 / (Altitudeh_d[lev + 1] - Altitudeh_d[lev])
                          * (KM_d[id * (nv + 1) + lev + 1] / (Altitude_d[lev + 1] - Altitude_d[lev])
                             + KM_d[id * (nv + 1) + lev] / Altitude_d[lev])
                      + 1.0 / time_step);
                ctmp[id * nv + lev] = KM_d[id * (nv + 1) + lev + 1]
                                      / ((Altitudeh_d[lev + 1] - Altitudeh_d[lev])
                                         * (Altitude_d[lev + 1] - Altitude_d[lev]));
                cpr_tmp[id * nv + lev] = ctmp[id * nv + lev] / btmp[id * nv + lev];
                for (int k = 0; k < 3; k++) {
                    dtmp[id * nv * 3 + lev * 3 + k] = -Mh_d[id * 3 * nv + lev * 3 + k] / time_step;
                    dpr_tmp[id * nv * 3 + lev * 3 + k] =
                        dtmp[id * nv * 3 + lev * 3 + k] / btmp[id * nv + lev];
                }
            }
            else if (lev == bl_top_lev_d[id]) {
                atmp[id * nv + lev] = KM_d[id * (nv + 1) + lev]
                                      / ((Altitudeh_d[lev + 1] - Altitudeh_d[lev])
                                         * (Altitude_d[lev] - Altitude_d[lev - 1]));
                btmp[id * nv + lev] =
                    -(1.0 / (Altitudeh_d[lev + 1] - Altitudeh_d[lev])
                          * (KM_d[id * (nv + 1) + lev + 1] / (Altitude_d[lev + 1] - Altitude_d[lev])
                             + KM_d[id * (nv + 1) + lev] / (Altitude_d[lev] - Altitude_d[lev - 1]))
                      + 1.0 / time_step);
                ctmp[id * nv + lev]    = 0;
                cpr_tmp[id * nv + lev] = 0; //not used, i think
                for (int k = 0; k < 3; k++) {
                    dtmp[id * nv * 3 + lev * 3 + k] =
                        -Mh_d[id * 3 * nv + lev * 3 + k] / time_step
                        - KM_d[id * (nv + 1) + lev + 1] / (Altitudeh_d[lev + 1] - Altitudeh_d[lev])
                              * Mh_d[id * 3 * nv + (lev + 1) * 3 + k]
                              / (Altitude_d[lev + 1] - Altitude_d[lev]);
                    dpr_tmp[id * nv * 3 + lev * 3 + k] =
                        (dtmp[id * nv * 3 + lev * 3 + k]
                         - atmp[id * nv + lev] * dpr_tmp[id * nv * 3 + (lev - 1) * 3 + k])
                        / (btmp[id * nv + lev] - atmp[id * nv + lev] * cpr_tmp[id * nv + lev - 1]);
                }
            }
            else {
                atmp[id * nv + lev] = KM_d[id * (nv + 1) + lev]
                                      / ((Altitudeh_d[lev + 1] - Altitudeh_d[lev])
                                         * (Altitude_d[lev] - Altitude_d[lev - 1]));
                btmp[id * nv + lev] =
                    -(1.0 / (Altitudeh_d[lev + 1] - Altitudeh_d[lev])
                          * (KM_d[id * (nv + 1) + lev + 1] / (Altitude_d[lev + 1] - Altitude_d[lev])
                             + KM_d[id * (nv + 1) + lev] / (Altitude_d[lev] - Altitude_d[lev - 1]))
                      + 1.0 / time_step);
                ctmp[id * nv + lev] = KM_d[id * (nv + 1) + lev + 1]
                                      / ((Altitudeh_d[lev + 1] - Altitudeh_d[lev])
                                         * (Altitude_d[lev + 1] - Altitude_d[lev]));
                cpr_tmp[id * nv + lev] =
                    ctmp[id * nv + lev]
                    / (btmp[id * nv + lev] - atmp[id * nv + lev] * cpr_tmp[id * nv + lev - 1]);
                for (int k = 0; k < 3; k++) {
                    dtmp[id * nv * 3 + lev * 3 + k] = -Mh_d[id * 3 * nv + lev * 3 + k] / time_step;
                    dpr_tmp[id * nv * 3 + lev * 3 + k] =
                        (dtmp[id * nv * 3 + lev * 3 + k]
                         - atmp[id * nv + lev] * dpr_tmp[id * nv * 3 + (lev - 1) * 3 + k])
                        / (btmp[id * nv + lev] - atmp[id * nv + lev] * cpr_tmp[id * nv + lev - 1]);
                }
            }
            if (fabs(btmp[id * nv + lev])
                < (fabs(atmp[id * nv + lev]) + fabs(ctmp[id * nv + lev]))) {
                printf("Warning! Thomas algorithm in boundary layer unstable\n");
            }
        }
        // if (id == 1000) {
        //     printf("stop");
        // }

        for (lev = bl_top_lev_d[id]; lev >= 0; lev--) {
            //backward sweep
            for (int k = 0; k < 3; k++) {
                if (lev == bl_top_lev_d[id]) {
                    Mh_d[id * nv * 3 + lev * 3 + k] = dpr_tmp[id * nv * 3 + lev * 3 + k];
                }
                else {
                    Mh_d[id * nv * 3 + lev * 3 + k] =
                        (dpr_tmp[id * nv * 3 + lev * 3 + k]
                         - cpr_tmp[id * nv + lev] * Mh_d[id * nv * 3 + (lev + 1) * 3 + k]);
                }
            }
        }
        // if (id == 0) {
        //     printf("%f\n", Mh_d[id * nv * 3 + 0]);
        // }
    }
}

__global__ void CalcRiB(double *pressure_d,
                        double *Rho_d,
                        double *Mh_d,
                        double *Tsurface_d,
                        double *Altitude_d,
                        double *Altitudeh_d,
                        double  Rd,
                        double  Cp,
                        double  P_Ref,
                        double  Gravit,
                        double *RiB_d,
                        int     num,
                        int     nv) {

    // Calculate bulk Richardson number for each level
    // The first value is defined at the midpoint between the lowest layer and the surface
    // The rest are at the interfaces between layers

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int lev;

    if (id < num) {
        double kappa = Rd / Cp;
        double p_surf, pt_surf, extrap_surf;
        double pt_layer, pt_lowest, pt_layer_below, pt_interface;
        double vh_layer, vh_layer_below, vh_interface;
        for (lev = 0; lev <= nv; lev++) {
            //first find surface pressure, and calculate pt at the interfaces

            if (lev == 0) {
                //lowest level, RiB defined at midpoint between lowest and surface
                // calculate pot temp of surface
                extrap_surf = -Altitude_d[lev + 1] / (Altitude_d[lev] - Altitude_d[lev + 1]);
                p_surf =
                    pressure_d[id * nv + lev + 1]
                    + extrap_surf * (pressure_d[id * nv + lev] - pressure_d[id * nv + lev + 1]);
                pt_surf = Tsurface_d[id] * pow(p_surf / P_Ref, -kappa);

                // calculate pt and horizontal velocity of layer
                pt_layer = pow(P_Ref, kappa) * pow(pressure_d[id * nv + lev], 1.0 - kappa)
                           / (Rho_d[id * nv + lev] * Rd);
                pt_lowest = pt_layer; //will need this later
                vh_layer  = sqrt((pow(Mh_d[id * nv * 3 + lev * 3 + 0], 2)
                                 + pow(Mh_d[id * nv * 3 + lev * 3 + 1], 2)
                                 + pow(Mh_d[id * nv * 3 + lev * 3 + 2], 2)))
                           / Rho_d[id * nv + lev];

                if (pow(vh_layer, 2) == 0) { //zero velocity, RiB = large +number
                    RiB_d[id * nv + lev] = LARGERiB;
                }
                else { // bulk Richardson number, wrt to surface
                    RiB_d[id * (nv + 1) + lev] = Gravit * Altitude_d[lev] * (pt_layer - pt_surf)
                                                 / (pt_surf * pow(vh_layer, 2));
                }
            }
            else if (lev == nv) {
                //what should I do at the top level??
                RiB_d[id * (nv + 1) + lev] = LARGERiB; //top level can't be incorporated into BL?
            }
            else {
                //potential temperatures for this layer, layer below, and interface b/w
                pt_layer_below = pt_layer;
                pt_layer       = pow(P_Ref, kappa) * pow(pressure_d[id * nv + lev], 1.0 - kappa)
                           / (Rho_d[id * nv + lev] * Rd);
                pt_interface = pt_layer_below
                               + (pt_layer - pt_layer_below)
                                     * (Altitudeh_d[lev] - Altitude_d[lev - 1])
                                     / (Altitude_d[lev] - Altitude_d[lev - 1]);

                //vh for the layers and interface
                vh_layer_below = vh_layer;
                vh_layer       = sqrt((pow(Mh_d[id * nv * 3 + lev * 3 + 0], 2)
                                 + pow(Mh_d[id * nv * 3 + lev * 3 + 1], 2)
                                 + pow(Mh_d[id * nv * 3 + lev * 3 + 2], 2)))
                           / Rho_d[id * nv + lev];
                vh_interface = vh_layer_below
                               + (vh_layer - vh_layer_below)
                                     * (Altitudeh_d[lev] - Altitude_d[lev - 1])
                                     / (Altitude_d[lev] - Altitude_d[lev - 1]);

                if (pow(vh_interface, 2) == 0) { //zero velocity, set RiB to a big +number
                    RiB_d[id * (nv + 1) + lev] = LARGERiB;
                }
                else { //bulk Ri number, wrt to lowest layer
                    RiB_d[id * (nv + 1) + lev] = Gravit * Altitudeh_d[lev]
                                                 * (pt_interface - pt_lowest)
                                                 / (pt_lowest * pow(vh_interface, 2));
                }
            }
        }
    }
}
